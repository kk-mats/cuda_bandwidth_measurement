﻿#include <iostream>

#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"


namespace kernel
{

__global__ void measure_global_bandwidth_kb(int *out, int *device, int size)
{
	int r=0;
	for(int i=0; i<size; ++i)
	{
		r+=device[i];
	}
	*out=r;
}

}


auto measure_host_device_bandwidth_mb(const int n, const bool enable_sync)
{
	const int bytes=n*(1<<20);

	int *host, *device;
	hipHostMalloc((void **)&host, bytes, hipHostMallocDefault);
	hipMalloc((void **)&device, bytes);

	hipEvent_t hd_start, hd_stop, dh_start, dh_stop;
	hipEventCreate(&hd_start);
	hipEventCreate(&hd_stop);
	hipEventCreate(&dh_start);
	hipEventCreate(&dh_stop);

	float hd_time, dh_time;

	if(enable_sync)
	{
		hipEventRecord(hd_start);
		hipMemcpy(device, host, bytes, hipMemcpyHostToDevice);
		hipEventRecord(hd_stop);
		hipEventSynchronize(hd_stop);

		hipEventRecord(dh_start);
		hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);
		hipEventRecord(dh_stop);
		hipEventSynchronize(dh_stop);
	}
	else
	{
		hipEventRecord(hd_start);
		hipMemcpyAsync(device, host, bytes, hipMemcpyHostToDevice);
		hipEventRecord(hd_stop);
		hipEventSynchronize(hd_stop);

		hipEventRecord(dh_start);
		hipMemcpyAsync(host, device, bytes, hipMemcpyDeviceToHost);
		hipEventRecord(dh_stop);
		hipEventSynchronize(dh_stop);
	}

	hipEventElapsedTime(&hd_time, hd_start, hd_stop);
	hipEventElapsedTime(&dh_time, dh_start, dh_stop);

	hipHostFree(host);
	hipFree(device);

	hipEventDestroy(hd_start);
	hipEventDestroy(hd_stop);
	hipEventDestroy(dh_start);
	hipEventDestroy(dh_stop);

	return std::make_pair(hd_time, dh_time);
}

void measure_host_device_bandwidth(const bool enable_sync)
{
	const int repeat=10;
	std::cout<<"host <-> device  "<<(enable_sync ? "sync" : "aysnc")<<std::endl;
	std::cout<<"data size[MB], host to device[ms], device to host[ms]"<<std::endl;
	for(int n=32; n<=256; n+=32)
	{
		float hd_sum=0, dh_sum=0;

		for(int i=0; i<repeat; ++i)
		{
			const auto time=measure_host_device_bandwidth_mb(n, enable_sync);
			hd_sum+=time.first;
			dh_sum+=time.second;
		}

		std::cout<<n<<", "<<hd_sum/repeat<<", "<<dh_sum/repeat<<std::endl;
	}
	std::cout<<"--\n"<<std::endl;
}

auto measure_global_bandwidth_kb(const int n)
{
	const int bytes=n*(1<<10)/2;
	
	int *out, *device;
	hipMalloc((void **)&out, sizeof(int));
	hipMalloc((void **)&device, bytes);

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 grid(1);
	dim3 threads(1);

	hipEventRecord(start);
	kernel::measure_global_bandwidth_kb<<<grid, threads>>>(out, device, bytes/sizeof(int));
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	
	hipFree(out);
	hipFree(device);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}

void measure_global_bandwidth()
{
	const int repeat=10;
	std::cout<<"global memory"<<std::endl;
	std::cout<<"data size[KB], time[ms]"<<std::endl;
	for(int n=32; n<=256; n+=32)
	{
		float sum=0;
		for(int i=0; i<repeat; ++i)
		{
			sum+=measure_global_bandwidth_kb(n);
		}

		std::cout<<n<<", "<<sum/repeat<<std::endl;
	}
	std::cout<<"--\n"<<std::endl;
}

int main()
{
	measure_host_device_bandwidth(true);
	measure_host_device_bandwidth(false);
	measure_global_bandwidth();
	return 0;
}

