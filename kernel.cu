﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <random>
#include <algorithm>

auto measure_host_to_device_memcopy_mb(const int n, const bool enable_async)
{
	constexpr long long mb=1<<20;
	const int size=n*mb/(2*sizeof(int));

	int *host;
	hipHostMalloc((void **)&host, size, hipHostMallocDefault);

	int *device;
	hipMalloc((void **)&device, size);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	if(!enable_async)
	{
		hipEventRecord(start);
		hipMemcpy(device, host, size, hipMemcpyHostToDevice);
		hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
		hipEventRecord(stop);
	}
	else
	{
		hipEventRecord(start);
		hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice);
		hipMemcpyAsync(host, device, size, hipMemcpyDeviceToHost);
		hipEventRecord(stop);
	}

	hipEventSynchronize(stop);

	hipHostFree(host);
	hipFree(device);

	float time=0;
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}


int main()
{
	for(const auto n :{16, 32, 64, 128, 256, 512, 1024, 2048})
	{
		float sum=0;

		for(int i=0; i<5; ++i)
		{
			const auto time=measure_host_to_device_memcopy_mb(n, false);
			std::cout<<"@sync_copy size: "<<n<<"MB, time="<<time<<std::endl;
			sum+=time;
		}

		std::cout<<"avg="<<sum/5<<"s"<<std::endl;
	}

	return 0;
}

